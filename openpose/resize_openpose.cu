#include "hip/hip_runtime.h"
// resize & norm for trt_pose

#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>

__forceinline__ __device__ float3 get(uchar3* src, int x, int y, int w, int h) 
{
	if (x < 0 || x >= w || y < 0 || y >= h) return make_float3(0.5, 0.5, 0.5);
	uchar3 temp = src[y*w + x];
// trt_pose range
	return make_float3(float(temp.x) / 255., float(temp.y) / 255., float(temp.z) / 255.);
}

__global__ void resizeNormKernel_openpose(uchar3* src, 
    float *dst, 
    int dstW, 
    int dstH, 
    int srcW, 
    int srcH,
	float scaleX, 
    float scaleY, 
    float shiftX, 
    float shiftY)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const int x = idx % dstW;
	int y = idx / dstW;
	if (x >= dstW || y >= dstH)
		return;
	float w = (x - shiftX + 0.5) * scaleX - 0.5;
	float h = (y - shiftY + 0.5) * scaleY - 0.5;
	int h_low = (int)h;
	int w_low = (int)w;
	int h_high = h_low + 1;
	int w_high = w_low + 1;
	float lh = h - h_low;
	float lw = w - w_low;
	float hh = 1.0 - lh, hw = 1.0 - lw;
	float w1 = hh * hw, w2 = hh * lw, w3 = lh * hw, w4 = lh * lw;
	float3 v1 = get(src, w_low, h_low, srcW, srcH);
	float3 v2 = get(src, w_high, h_low, srcW, srcH);
	float3 v3 = get(src, w_low, h_high, srcW, srcH);
	float3 v4 = get(src, w_high, h_high, srcW, srcH);
	int stride = dstW * dstH;
//	dst[y*dstW + x] = w1 * v1.x + w2 * v2.x + w3 * v3.x + w4 * v4.x;
//	dst[stride + y * dstW + x] = w1 * v1.y + w2 * v2.y + w3 * v3.y + w4 * v4.y;
//	dst[stride * 2 + y * dstW + x] = w1 * v1.z + w2 * v2.z + w3 * v3.z + w4 * v4.z;



	float r = w1 * v1.x + w2 * v2.x + w3 * v3.x + w4 * v4.x;
	float g = w1 * v1.y + w2 * v2.y + w3 * v3.y + w4 * v4.y;
	float b = w1 * v1.z + w2 * v2.z + w3 * v3.z + w4 * v4.z;


// extra step which improves performance in trt_pose
// image.sub_(mean[:, None, None]).div_(std[:, None, None])
//      r -= 0.485;
//      g -= 0.456;
//      b -= 0.406;
//      r /= 0.229;
//      g /= 0.224;
//      b /= 0.225;


    dst[y*dstW + x] = r;
    dst[stride + y * dstW + x] = g;
    dst[stride * 2 + y * dstW + x] = b;
}

// Decompression coefficients straight out of jpeglib
#define V_TO_R    1.40200
#define V_TO_G    -0.71414
#define U_TO_G    -0.34414
#define U_TO_B    1.77200


__device__ float3 getYUV(uchar1* src, 
    int x, 
    int y, 
    int w, 
    int h) 
{
//	if (x < 0 || x >= w || y < 0 || y >= h) return make_float3(0.0, 0.0, 0.0);
    float3 yuv;
    int offset_uv = y * w * 2 + x / 2 * 4;
	yuv.x = float(src[y * w * 2 + x * 2].x) / 255;
    yuv.y = float(src[offset_uv + 1].x) / 255 - 0.5;
    yuv.z = float(src[offset_uv + 3].x) / 255 - 0.5;

    float3 rgb;
	rgb.x = yuv.x + V_TO_R * yuv.z;
	rgb.y = yuv.x + U_TO_G * yuv.y + V_TO_G * yuv.z;
	rgb.z = yuv.x + U_TO_B * yuv.y;
    return rgb;
}

__global__ void resizeNormKernel_landscape(uchar1* src, 
    float *dst, 
    int dstW, 
    int dstH, 
    int srcW, 
    int srcH,
	float scaleX, 
    float scaleY)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const int x = idx % dstW;
	int y = idx / dstW;
	if (x >= dstW || y >= dstH)
		return;
	float w = (x + 0.5) * scaleX - 0.5;
	float h = (y + 0.5) * scaleY - 0.5;
	int h_low = (int)h;
	int w_low = (int)w;
	int h_high = h_low + 1;
	int w_high = w_low + 1;
	float lh = h - h_low;
	float lw = w - w_low;
	float hh = 1.0 - lh, hw = 1.0 - lw;
	float w1 = hh * hw, w2 = hh * lw, w3 = lh * hw, w4 = lh * lw;
	float3 v1 = getYUV(src, w_low, h_low, srcW, srcH);
	float3 v2 = getYUV(src, w_high, h_low, srcW, srcH);
	float3 v3 = getYUV(src, w_low, h_high, srcW, srcH);
	float3 v4 = getYUV(src, w_high, h_high, srcW, srcH);
	int stride = dstW * dstH;

	float r = w1 * v1.x + w2 * v2.x + w3 * v3.x + w4 * v4.x;
	float g = w1 * v1.y + w2 * v2.y + w3 * v3.y + w4 * v4.y;
	float b = w1 * v1.z + w2 * v2.z + w3 * v3.z + w4 * v4.z;


    dst[y*dstW + x] = r;
    dst[stride + y * dstW + x] = g;
    dst[stride * 2 + y * dstW + x] = b;
}

__global__ void resizeNormKernel_portrait(uchar1* src, 
    float *dst, 
    int dstW, 
    int dstH, 
    int srcW, 
    int srcH,
	float scaleX, 
    float scaleY, 
    float shiftX, 
    float shiftY)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	const int x = idx % dstW;
	int y = idx / dstW;
	if (x >= dstW || y >= dstH)
		return;
// rotate & scale
	float w = (-y + 0.5) * scaleX + shiftX - 0.5;
	float h = (x + 0.5) * scaleY - shiftY - 0.5;
	int h_low = (int)h;
	int w_low = (int)w;
	int h_high = h_low + 1;
	int w_high = w_low + 1;
	float lh = h - h_low;
	float lw = w - w_low;
	float hh = 1.0 - lh, hw = 1.0 - lw;
	float w1 = hh * hw, w2 = hh * lw, w3 = lh * hw, w4 = lh * lw;
	float3 v1 = getYUV(src, w_low, h_low, srcW, srcH);
	float3 v2 = getYUV(src, w_high, h_low, srcW, srcH);
	float3 v3 = getYUV(src, w_low, h_high, srcW, srcH);
	float3 v4 = getYUV(src, w_high, h_high, srcW, srcH);
	int stride = dstW * dstH;

	float r = w1 * v1.x + w2 * v2.x + w3 * v3.x + w4 * v4.x;
	float g = w1 * v1.y + w2 * v2.y + w3 * v3.y + w4 * v4.y;
	float b = w1 * v1.z + w2 * v2.z + w3 * v3.z + w4 * v4.z;


    dst[y*dstW + x] = r;
    dst[stride + y * dstW + x] = g;
    dst[stride * 2 + y * dstW + x] = b;
}

int resizeAndNorm_yuv(void *src, 
    float *dst, 
    int src_w, 
    int src_h, 
    int dst_w, 
    int dst_h, 
    hipStream_t stream) 
{
	float scaleX;
	float scaleY;
	float shiftX = 0.f;
    float shiftY = 0.f;

	const int n = dst_w * dst_h;
	int blockSize = 512;
	const int gridSize = (n + blockSize - 1) / blockSize;

    if(dst_w > dst_h)
    {
// landscape
    	scaleX = src_w * 1.0f / dst_w;
	    scaleY = src_h * 1.0f / dst_h;
        resizeNormKernel_landscape << <gridSize, blockSize, 0, stream >> > ((uchar1*)(src), dst, dst_w, dst_h, src_w, src_h, scaleX, scaleY);
    }
    else
    {
// portrait
// camera cropping for 3:2
        float src_w2 = src_w * 540 / 640;
        scaleX = src_w2 * 1.0f / dst_h;
        scaleY = src_h * 1.0f / dst_w;
        shiftX = (src_w2 - src_w) / 2;
        resizeNormKernel_portrait << <gridSize, blockSize, 0, stream >> > ((uchar1*)(src), dst, dst_w, dst_h, src_w, src_h, scaleX, scaleY, shiftX, shiftY);
// printf("resizeAndNorm_yuv %d src=%dx%d dst=%dx%d scaleX=%f scaleY=%f shiftX=%f\n",
// __LINE__,
// src_w,
// src_h,
// dst_w,
// dst_h,
// scaleX,
// scaleY,
// shiftX);
    }


//printf("resizeAndNorm %d %f %f %f %f\n", __LINE__, scaleX, scaleY, shiftX, shiftY);
	return 0;
}








